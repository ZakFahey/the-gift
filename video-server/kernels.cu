#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "utils.h"

const int VID_WIDTH=1920, VID_HEIGHT=1080, FACE_WIDTH=256, FACE_HEIGHT=384;
const int BLOCK_SIZE=256;

__global__ void interpolate(int* keyFrames, int* frameData, int numKeyframes, int numFrames){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	if(idx<numFrames){
		int startFrame=0;
		for(int i=1; i<=numKeyframes; i++){
			if(idx<=(int)keyFrames[i*3]){
				startFrame=i-1;
				break;
			}
		}
		int t=idx-keyFrames[startFrame*3];
		int dt=keyFrames[(startFrame+1)*3]-keyFrames[startFrame*3];
		int dx=keyFrames[(startFrame+1)*3+1]-keyFrames[startFrame*3+1];
		int dy=keyFrames[(startFrame+1)*3+2]-keyFrames[startFrame*3+2];
//		printf("t: %i\tdt: %i\tdx: %i\tvx: %i\tdy: %i\tvy: %i\n", t, dt, dx, dx*t/dt, dy, dy*t/dt);
		frameData[idx*2]=keyFrames[startFrame*3+1]+dx*t/dt;
		frameData[idx*2+1]=keyFrames[startFrame*3+2]+dy*t/dt;
	}
	__syncthreads();
}

__global__ void processFace(uchar3* face){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	if(idx<FACE_WIDTH*FACE_HEIGHT){
		int x=idx%FACE_WIDTH;
		int y=idx/FACE_WIDTH;
		int xc=FACE_WIDTH/2, yc=FACE_HEIGHT/2;
//		printf("x: %i\ty: %i\txc: %i\tyc: %i\tx/a: %f\ty/b: %f\n", x, y, xc, yc, (float)((x-xc)*(x-xc))/(xc*xc), (float)((y-yc)*(y-yc))/(yc*yc));
//		printf("dist: %i\n", ((x-xc)*(x-xc))/(xc*xc)+((y-yc)*(y-yc))/(yc*yc));
		if((float)((x-xc)*(x-xc))/(xc*xc)+(float)((y-yc)*(y-yc))/(yc*yc)>1){
			face[idx].x=0;
			face[idx].y=0;
			face[idx].z=0;
		}
	}
	__syncthreads();
}

__global__ void processFrame(uchar3* face, uchar3* frame, int xOffset, int yOffset, int xc, int yc, int fxc, int fyc){
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	if(idx<FACE_WIDTH*FACE_HEIGHT && face[idx].x!=0 && face[idx].y!=0 && face[idx].z!=0){
		int face_x=idx%FACE_WIDTH;
		int face_y=idx/FACE_WIDTH;
		int frame_x=xc+xOffset+face_x-fxc;
		int frame_y=yc-yOffset+face_y-fyc;
		if(frame_x<VID_WIDTH && frame_x>=0 && frame_y<VID_HEIGHT && frame_y>=0 && face[face_x+face_y*FACE_WIDTH].x!=0 && face[face_x+face_y*FACE_WIDTH].y!=0 && face[face_x+face_y*FACE_WIDTH].z!=0){
			frame[frame_x+frame_y*VID_WIDTH].x=face[face_x+face_y*FACE_WIDTH].x;
			frame[frame_x+frame_y*VID_WIDTH].y=face[face_x+face_y*FACE_WIDTH].y;
			frame[frame_x+frame_y*VID_WIDTH].z=face[face_x+face_y*FACE_WIDTH].z;
		}
	}
	__syncthreads();
}

__global__ void processFrames(uchar3* face, uchar3* video, int* frameData, int numFrames, int xc, int yc, int fxc, int fyc){
	int idxA=threadIdx.x+blockIdx.x*blockDim.x;
	int idxB=threadIdx.y+blockIdx.y*blockDim.y;
	int idx=idxA+idxB*BLOCK_SIZE;
	int idx2=idx%(FACE_WIDTH*FACE_HEIGHT);
	if(idx<FACE_WIDTH*FACE_HEIGHT*numFrames && face[idx2].x!=0 && face[idx2].y!=0 && face[idx2].z!=0){
		int frame=idx/(FACE_WIDTH*FACE_HEIGHT);
		int face_x=idx2%FACE_WIDTH;
		int face_y=idx2/FACE_WIDTH;
		int frame_x=xc+frameData[idx*2]+face_x-fxc;
		int frame_y=yc-frameData[idx*2+1]+face_y-fyc;
		if(frame_x<VID_WIDTH && frame_x>=0 && frame_y<VID_HEIGHT && frame_y>=0){
			video[frame_x+frame_y*VID_WIDTH+VID_WIDTH*VID_HEIGHT*frame].x=face[idx2].x;
			video[frame_x+frame_y*VID_WIDTH+VID_WIDTH*VID_HEIGHT*frame].y=face[idx2].y;
			video[frame_x+frame_y*VID_WIDTH+VID_WIDTH*VID_HEIGHT*frame].z=face[idx2].z;
		}
	}
	__syncthreads();
}

void d_interpolate(int* h_keyFrames, int* h_frameData, int numKeyframes, int numFrames){
	thrust::device_vector<int> d_keyFrames(h_keyFrames, h_keyFrames+numKeyframes*3);
	thrust::device_vector<int> d_frameData(h_frameData, h_frameData+numFrames*3);
	interpolate<<<numFrames/BLOCK_SIZE+1, BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_keyFrames[0]), thrust::raw_pointer_cast(&d_frameData[0]), numKeyframes, numFrames);
	thrust::copy(d_frameData.begin(), d_frameData.end(), h_frameData);
}

void d_processFace(uchar3* h_face){
	thrust::device_vector<uchar3> d_face(h_face, h_face+FACE_WIDTH*FACE_HEIGHT);
	processFace<<<FACE_WIDTH*FACE_HEIGHT/BLOCK_SIZE, BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_face[0]));
	thrust::copy(d_face.begin(), d_face.end(), h_face);
}

void d_processFrame(uchar3* h_face, uchar3* h_frame, int xOffset, int yOffset){
	thrust::device_vector<uchar3> d_face(h_face, h_face+FACE_WIDTH*FACE_HEIGHT);
	thrust::device_vector<uchar3> d_frame(h_frame, h_frame+VID_WIDTH*VID_HEIGHT);
	processFrame<<<FACE_WIDTH*FACE_HEIGHT/BLOCK_SIZE, BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_face[0]), thrust::raw_pointer_cast(&d_frame[0]), xOffset, yOffset, VID_WIDTH/2, VID_HEIGHT/2, FACE_WIDTH/2, FACE_HEIGHT/2);
	thrust::copy(d_frame.begin(), d_frame.end(), h_frame);
}

void d_processFrames(uchar3* h_face, uchar3* h_video, int* h_frameData, int numFrames){
	thrust::device_vector<uchar3> d_face(h_face, h_face+FACE_WIDTH*FACE_HEIGHT);
	printf("face copied\n");
	thrust::device_vector<uchar3> d_video(h_video, h_video+VID_WIDTH*VID_HEIGHT*numFrames);
	printf("video copied\n");
	thrust::device_vector<int> d_frameData(h_frameData, h_frameData+numFrames*2);
	printf("frames copied\n");
	int dim=sqrt(VID_HEIGHT*VID_WIDTH*numFrames)+1;
	dim3 BLOCK2D(BLOCK_SIZE, BLOCK_SIZE);
	dim3 NUMBLOCKS2D(dim/BLOCK_SIZE, dim/BLOCK_SIZE);
//	processFrames<<<FACE_WIDTH*FACE_HEIGHT/BLOCK_SIZE*numFrames, BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_face[0]), thrust::raw_pointer_cast(&d_video[0]), thrust::raw_pointer_cast(&d_frameData[0]), numFrames, VID_WIDTH/2, VID_HEIGHT/2, FACE_WIDTH/2, FACE_HEIGHT/2);
	processFrames<<<NUMBLOCKS2D, BLOCK2D>>>(thrust::raw_pointer_cast(&d_face[0]), thrust::raw_pointer_cast(&d_video[0]), thrust::raw_pointer_cast(&d_frameData[0]), numFrames, VID_WIDTH/2, VID_HEIGHT/2, FACE_WIDTH/2, FACE_HEIGHT/2);
	printf("video processed\n");
	thrust::copy(d_video.begin(), d_video.end(), h_video);
	printf("video copied back\n");
}
